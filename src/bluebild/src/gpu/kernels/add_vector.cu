#include "hip/hip_runtime.h"
#include <algorithm>
#include "bluebild//config.h"
#include "gpu/util/gpu_runtime.hpp"
#include "gpu/util/gpu_runtime_api.hpp"
#include "gpu/kernels/add_vector.hpp"

namespace bluebild {
namespace gpu {

template <typename T>
__global__ void
add_vector_kernel(int n, const gpu::ComplexType<T> *__restrict__ a, T *b) {
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
       i += gridDim.x * blockDim.x) {
    b[i] += a[i].x;
  }
}

template <typename T>
auto add_vector(gpu::StreamType stream, int n, const gpu::ComplexType<T> *a,
                T *b) -> void {
  constexpr int blockSize = 256;
  constexpr int maxBlocks = 65535;

  dim3 block(blockSize, 1, 1);
  dim3 grid(std::min<unsigned int>(maxBlocks, (n + block.x - 1) / block.x), 1,
            1);
  gpu::launch_kernel(add_vector_kernel<T>, grid, block, 0, stream, n, a, b);
}

template auto add_vector<float>(gpu::StreamType stream, int n,
                                const gpu::ComplexType<float> *a, float *b)
    -> void;

template auto add_vector<double>(gpu::StreamType stream, int n,
                                 const gpu::ComplexType<double> *a, double *b)
    -> void;

} // namespace gpu
} // namespace bluebild
